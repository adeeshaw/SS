#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Simple example demonstrating how to use MPI with CUDA
 *
 *  Generate some random numbers on one node.
 *  Dispatch them to all nodes.
 *  Compute their square root on each node's GPU.
 *  Compute the average of the results using MPI.
 *
 *  simpleMPI.cpp: main program, compiled with mpicxx on linux/Mac platforms
 *                 on Windows, please download the Microsoft HPC Pack SDK 2008
 */

// System includes
#include <iostream>

using std::cout;
using std::cerr;
using std::endl;



#include <iostream>
using std::cerr;
using std::endl;

#include <stdio.h>
#include <hip/hip_complex.h>
#include <complex.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <math.h>

#include "simpleMPI.h"



// Error handling macro
/*#define CUDA_CHECK(call) \
  if((call) != hipSuccess) { \
  hipError_t err = hipGetLastError(); \
  cerr << "CUDA error calling \""#call"\", code is " << err << endl; \
  my_abort(err); }
 */


// Device code
// Very simple GPU Kernel that computes square roots of input numbers

__global__ void BPKernel(int gpu_id,int num_gpus, float2 *rc, float *im_final_R, float *im_final_Q,
		int Nx,int Ny, int Nfft,  float4 *AntXYZR, float minF, int Np, int Wx, int Wy, int x0,
		int y0, float prepi, float prep2, float prep3,int multiresarray_size,int sar_data_multiresarray_offset_size, int *multiresarray,int gridwidth, int totalmpistreamtimes,int mpistreamtimes) {
	//Np=1;
	int x=0, index=0;
	float2 y_lo_first, y_hi_first;


	if( blockIdx.y* gridwidth + (blockIdx.x) < multiresarray_size ) {


		//	clock_t startset, endset;

		//	startset = clock();

		if (multiresarray[(blockIdx.y*gridwidth +blockIdx.x)*5+4] == 2) {
#pragma unroll
			for (int threadloop = 0; threadloop < BOX_SIZEX * BOX_SIZEY;
					threadloop++) {
				float2 y_lo_first, y_hi_first;

				float dR =0, tmpRA=0, final_imvalueR=0, final_imvalueQ=0, 
				      phCorrR=0, phCorrQ=0, getinterpR=0, getinterpQ=0, x_lo=0,
				      y_lo=0, y_hi=0, imag_y_lo=0, imag_y_hi=0,x_mat=0, y_mat=0,
				      tempdx=0;

				x_mat = (x0 - Wx / 2)
					+ ( multiresarray[ (blockIdx.y*gridwidth +blockIdx.x)*5+0] + (threadIdx.x)* BOX_SIZEX+ threadloop%BOX_SIZEX)
					* Wx / (float) (Nx - 1);


				y_mat = (y0 - Wy / 2)
					+ (multiresarray[(blockIdx.y*gridwidth +blockIdx.x)*5+1]+ (threadIdx.y)* BOX_SIZEY +threadloop/BOX_SIZEY ) 
					* Wy / (float) (Ny - 1);

				//if(blockIdx.y==1&& blockIdx.x==1&& threadIdx.x==0&& threadIdx.y==0 ){

				//	printf("%d %.40f %.40f \n", threadloop, x_mat, y_mat);

				//	}


				for (x = (Np/totalmpistreamtimes)*mpistreamtimes; x < (Np/totalmpistreamtimes)*mpistreamtimes+ Np/totalmpistreamtimes; x++) {

					tmpRA = AntXYZR[x].x  - x_mat;
					dR = tmpRA * tmpRA;
					tmpRA = AntXYZR[x].y - y_mat;
					dR += tmpRA * tmpRA;
					tmpRA = AntXYZR[x].z;
					dR += tmpRA * tmpRA;



					dR = sqrt(dR) - AntXYZR[x].w;

					tmpRA = prepi * minF * dR;

					__sincosf(tmpRA, &phCorrQ, &phCorrR  );


					index = dR * prep3 + Nfft/2;



					x_lo = (-Nfft / 2 + index) * prep2;



					y_lo_first = rc[x*Nfft+index];
					y_hi_first = rc[x*Nfft+index+ 1];


					y_lo = y_lo_first.x;
					y_hi = y_hi_first.x;

					imag_y_lo = y_lo_first.y;
					imag_y_hi = y_hi_first.y;

					tempdx =((dR - x_lo) / prep2);


					getinterpR = y_lo + tempdx * (y_hi - y_lo);

					//	y_lo = 11.3;
					//	y_hi = 11.5;


					getinterpQ = imag_y_lo
						+ tempdx * (imag_y_hi - imag_y_lo);


					final_imvalueR += ((getinterpR * phCorrR) - (getinterpQ * phCorrQ));
					final_imvalueQ += ((getinterpR * phCorrQ) + (getinterpQ * phCorrR));

				}


				im_final_R[ (blockIdx.y*gridwidth +blockIdx.x)*RES_SIZEY*RES_SIZEX+ (threadIdx.y*BLOCK_SIZEX+threadIdx.x)*BOX_SIZEY*BOX_SIZEX +threadloop] += final_imvalueR;

				im_final_Q[ (blockIdx.y*gridwidth +blockIdx.x)*RES_SIZEY*RES_SIZEX+ (threadIdx.y*BLOCK_SIZEX+threadIdx.x)*BOX_SIZEY*BOX_SIZEX +threadloop] += final_imvalueQ;

				//printf("inside value %.40f \n", final_imvalueR);

				/*im_final_R[ (blockIdx.y*gridwidth +blockIdx.x)*RES_SIZEX*RES_SIZEY+ threadIdx.y+threadIdx.x] +=final_imvalueR;
				  im_final_Q[ (blockIdx.y*gridwidth +blockIdx.x)*RES_SIZEX*RES_SIZEY+ threadIdx.y+threadIdx.x]  += final_imvalueQ;*/



				//im_final_R[ (blockIdx.y*gridwidth +blockIdx.x)*RES_SIZEX*RES_SIZEY+ (threadIdx.y*RES_SIZEX+threadIdx.x*BOX_SIZEX*BOX_SIZEY)+threadloop] +=final_imvalueR;
				//im_final_Q[ (blockIdx.y*gridwidth +blockIdx.x)*RES_SIZEX*RES_SIZEY+ (threadIdx.y*RES_SIZEX+threadIdx.x*BOX_SIZEX*BOX_SIZEY)+threadloop]  += final_imvalueQ;


			}



		}


		else if (multiresarray[(blockIdx.y*gridwidth +blockIdx.x)*5+4] == 1) {


			//printf("this cannot print tadaaaaaaaaaaaaaaa\n");
			for (int threadloop = 0; threadloop < BOX_SIZEX * BOX_SIZEY/4;
					threadloop++) {

				float dR =0, tmpRA=0, final_imvalueR=0, final_imvalueQ=0, 
				      phCorrR=0, phCorrQ=0, getinterpR=0, getinterpQ=0, x_lo=0,
				      y_lo=0, y_hi=0, imag_y_lo=0, imag_y_hi=0,x_mat=0, y_mat=0,
				      tempdx=0;

				x_mat = (x0 - Wx / 2)
					+ ( multiresarray[ (blockIdx.y*gridwidth +blockIdx.x)*5+0] + (threadIdx.x)* BOX_SIZEX+ 2*(threadloop%(BOX_SIZEX/2)))
					* Wx / (float) (Nx - 1);


				y_mat = (y0 - Wy / 2)
					+ (multiresarray[(blockIdx.y*gridwidth +blockIdx.x)*5+1]+ (threadIdx.y)* BOX_SIZEY +2*(threadloop/(BOX_SIZEY/2)) ) 
					* Wy / (float) (Ny - 1);



				//if(currentvalue==0){
				//		printf("%s %d  \n", "change happens : ", currentvalue);

				//		printf("%d %.40f %.40f \n", currentvalue, x_mat, y_mat);

				//	}


				for (x = (Np/totalmpistreamtimes)*mpistreamtimes; x < (Np/totalmpistreamtimes)*mpistreamtimes+ Np/totalmpistreamtimes; x++) {

					tmpRA = AntXYZR[x].x  - x_mat;
					dR = tmpRA * tmpRA;
					tmpRA = AntXYZR[x].y - y_mat;
					dR += tmpRA * tmpRA;
					tmpRA = AntXYZR[x].z;
					dR += tmpRA * tmpRA;



					dR = sqrt(dR) - AntXYZR[x].w;

					tmpRA = prepi * minF * dR;

					__sincosf(tmpRA, &phCorrQ, &phCorrR  );


					index = dR * prep3 + Nfft/2;



					x_lo = (-Nfft / 2 + index) * prep2;



					y_lo_first = rc[x*Nfft+index];
					y_hi_first = rc[x*Nfft+index+ 1];


					y_lo = y_lo_first.x;
					y_hi = y_hi_first.x;

					imag_y_lo = y_lo_first.y;
					imag_y_hi = y_hi_first.y;

					tempdx =((dR - x_lo) / prep2);


					getinterpR = y_lo + tempdx * (y_hi - y_lo);

					//	y_lo = 11.3;
					//	y_hi = 11.5;


					getinterpQ = imag_y_lo
						+ tempdx * (imag_y_hi - imag_y_lo);


					final_imvalueR += ((getinterpR * phCorrR) - (getinterpQ * phCorrQ));
					final_imvalueQ += ((getinterpR * phCorrQ) + (getinterpQ * phCorrR));



				}


				for(int fillx = 0; fillx < BOX_SIZEX; fillx++) {

					im_final_R[ (blockIdx.y*gridwidth +blockIdx.x)*RES_SIZEY*RES_SIZEX+ (threadIdx.y*BLOCK_SIZEX+threadIdx.x)*BOX_SIZEX*BOX_SIZEY +threadloop*BOX_SIZEX+ fillx] += final_imvalueR;
					im_final_Q[ (blockIdx.y*gridwidth +blockIdx.x)*RES_SIZEY*RES_SIZEX+ (threadIdx.y*BLOCK_SIZEX+threadIdx.x)*BOX_SIZEX*BOX_SIZEY +threadloop*BOX_SIZEX+ fillx] += final_imvalueQ;



				}


				//im_final_R[ (bl ckIdx.y*gridwidth +blockIdx.x)*RES_SIZE*RES_SIZE+ (threadIdx.y*BLOCK_SIZE+threadIdx.x)*BOX_SIZE*BOX_SIZE +threadloop] = final_imvalueR;
				//	im_final_Q[(currentvalue + ((threadloop / BOX_SIZE) * Nx)
				//		+ (threadloop % BOX_SIZE)) - (sar_data_multiresarray_offset_size*RES_SIZE*RES_SIZE)] = final_imvalueQ;



				//im_final_R[currentvalue+threadloop] = final_imvalueR;
				//im_final_Q[currentvalue+threadloop] = final_imvalueQ;

				//if(gpu_id==1 &&(currentvalue>8388350|| currentvalue< 4194310)){

				//	printf("%d %.40f %.40f \n", currentvalue, final_imvalueR, final_imvalueQ);

				//}

			}



		}
		else if(multiresarray[(blockIdx.y*gridwidth +blockIdx.x)*5+4] == 0) {
			//	printf("this cannot print tadaaaaaaaaaaaaaaa\n");
			//if(currentvalue<512){

			//printf("%d dddddddddddddddddddddddddd  \n", currentvalue);

			//}

			float dR =0, tmpRA=0, final_imvalueR=0, final_imvalueQ=0, 
			      phCorrR=0, phCorrQ=0, getinterpR=0, getinterpQ=0, x_lo=0,
			      y_lo=0, y_hi=0, imag_y_lo=0, imag_y_hi=0,x_mat=0, y_mat=0,
			      tempdx=0;

			x_mat = (x0 - Wx / 2)
				+ ( multiresarray[ (blockIdx.y*gridwidth +blockIdx.x)*5+0] + (threadIdx.x)* BOX_SIZEX)
				* Wx / (float) (Nx - 1);


			y_mat = (y0 - Wy / 2)
				+ ((threadIdx.y)* BOX_SIZEY + multiresarray[(blockIdx.y*gridwidth +blockIdx.x)*5+1]) 
				* Wy / (float) (Ny - 1);




			for (x = (Np/totalmpistreamtimes)*mpistreamtimes; x < (Np/totalmpistreamtimes)*mpistreamtimes+ Np/totalmpistreamtimes; x++) {

				tmpRA = AntXYZR[x].x  - x_mat;
				dR = tmpRA * tmpRA;
				tmpRA = AntXYZR[x].y - y_mat;
				dR += tmpRA * tmpRA;
				tmpRA = AntXYZR[x].z;
				dR += tmpRA * tmpRA;



				dR = sqrt(dR) - AntXYZR[x].w;

				tmpRA = prepi * minF * dR;

				__sincosf(tmpRA, &phCorrQ, &phCorrR  );


				index = dR * prep3 + Nfft/2;



				x_lo = (-Nfft / 2 + index) * prep2;



				y_lo_first = rc[x*Nfft+index];
				y_hi_first = rc[x*Nfft+index+ 1];


				y_lo = y_lo_first.x;
				y_hi = y_hi_first.x;

				imag_y_lo = y_lo_first.y;
				imag_y_hi = y_hi_first.y;

				tempdx =((dR - x_lo) / prep2);


				getinterpR = y_lo + tempdx * (y_hi - y_lo);

				//	y_lo = 11.3;
				//	y_hi = 11.5;


				getinterpQ = imag_y_lo
					+ tempdx * (imag_y_hi - imag_y_lo);


				final_imvalueR += ((getinterpR * phCorrR) - (getinterpQ * phCorrQ));
				final_imvalueQ += ((getinterpR * phCorrQ) + (getinterpQ * phCorrR));


			}



			for (int fillx = 0; fillx < BOX_SIZEX*BOX_SIZEX; fillx++) {

				im_final_R[ (blockIdx.y*gridwidth +blockIdx.x)*RES_SIZEX*RES_SIZEY+ (threadIdx.y*BLOCK_SIZEX+threadIdx.x)*BOX_SIZEY*BOX_SIZEX + fillx] +=final_imvalueR;
				im_final_Q[ (blockIdx.y*gridwidth +blockIdx.x)*RES_SIZEX*RES_SIZEY+ (threadIdx.y*BLOCK_SIZEX+threadIdx.x)*BOX_SIZEY*BOX_SIZEX + fillx] +=final_imvalueQ;
			}




		} 







		/*for(int k=0;k<16;k++){

		  im_final_R[ (blockIdx.y*gridwidth +blockIdx.x)*RES_SIZEX*RES_SIZEY+ (threadIdx.y*BLOCK_SIZEX*1+threadIdx.x*16) +k] = final_imvalueR[k];
		  im_final_Q[ (blockIdx.y*gridwidth +blockIdx.x)*RES_SIZEX*RES_SIZEY+ (threadIdx.y*BLOCK_SIZEX*1+threadIdx.x*16) +k] = final_imvalueQ[k];

		  } */


		//	endset = clock();

		//	printf("Number of seconds:  %.10f threadid %d\n", (endset - startset) / (double) CLOCKS_PER_SEC, threadIdx.x);


	}


}

void GPU_Init(int gpu_id){

	hipError_t ierr_cuda;



		ierr_cuda= hipSetDevice(7);
		// modified to run on 100 cpu machine
		//ierr_cuda= hipSetDevice(0);


		if (hipGetLastError() != hipSuccess) {
			printf("CUDA error set device: %s\n", hipGetErrorString(ierr_cuda));
			exit(0);
		}

}

	clock_t startset, endset;
	float2* rc;
float4* sar_data_Antxyzr;
float* d_im_final_R;
float* d_im_final_Q;
float4* Antxyzr_d;
int* multiresarray_d;
float2* d_rc;


void bpbasic(int gpu_id,int num_gpus, int sar_data_Np, int sar_data_Nfft, int sar_data_Nx,int sar_data_Ny,float sar_data_minF,float* sar_data_xyzr, float* sar_data_finalIfft_RQ, int sar_data_multiresarray_size, int sar_data_multiresarray_offset_size, int* sar_data_multiresarray,int sar_data_K, int sar_data_Wx, int sar_data_Wy, int sar_data_x0, int sar_data_y0,   float prep2, float prep3, float prepi,float *im_final_host_R,float *im_final_host_Q,int totalmpistreamtimes,int mpistreamtimes1) {

	//printf("GPU ID _ %d \n", gpu_id);

	hipEvent_t start9, stop9;
	hipEventCreate(&start9);
	hipEventCreate(&stop9);
	hipEventRecord(start9);



	//if(mpistreamtimes==0){
		

		
		//hipError_t ierr_cuda;



		// Do something

		//ierr_cuda= hipSetDevice(gpu_id);
		// modified to run on 100 cpu machine
		//ierr_cuda= hipSetDevice(0);


		//endset = clock();

		//printf("Number of seconds:  %.10f GPU %d\n", (endset - startset) / (double) CLOCKS_PER_SEC, gpu_id);



		//if (hipGetLastError() != hipSuccess) {
		//	printf("CUDA error set device: %s\n", hipGetErrorString(ierr_cuda));
		//	exit(0);
		//}
		hipEvent_t start8, stop8;
	hipEventCreate(&start8);
	hipEventCreate(&stop8);
	hipEventRecord(start8);

		rc = (float2*) sar_data_finalIfft_RQ ; 

		sar_data_Antxyzr = (float4*) sar_data_xyzr;



		hipMalloc(&d_im_final_R, sizeof(float) * (sar_data_multiresarray_size*RES_SIZEX*RES_SIZEY));
		//printf("point3Cuda Done\n");
		if (hipGetLastError() != hipSuccess) {
			printf("point3Cuda error: Failed to allocate\n");
			//exit(0);
		}



		hipMalloc(&d_im_final_Q, sizeof(float) * (sar_data_multiresarray_size*RES_SIZEX*RES_SIZEY));

		hipMemset(d_im_final_R, 0.0, sizeof(float) * (sar_data_multiresarray_size*RES_SIZEX*RES_SIZEY));
		hipMemset(d_im_final_Q, 0.0, sizeof(float) * (sar_data_multiresarray_size*RES_SIZEX*RES_SIZEY));


		hipMalloc(&Antxyzr_d, sizeof(float) * sar_data_Np*4);
		//printf("point4Cuda Done\n");
		if (hipGetLastError() != hipSuccess) {
			printf("point4Cuda error: Failed to allocate\n");
			//exit(0);
		}


		hipMalloc(&multiresarray_d, sizeof(int) * sar_data_multiresarray_size*5);

		//printf("point11Cuda Done\n");
		if (hipGetLastError() != hipSuccess) {
			printf("point11Cuda error: Failed to allocate\n");
			//exit(0);
		}

		hipMemcpy(multiresarray_d, sar_data_multiresarray,
				sizeof(int) * sar_data_multiresarray_size*5,
				hipMemcpyHostToDevice);
		//printf("point93Cuda Done\n");
		if (hipGetLastError() != hipSuccess) {
			printf("point93Cuda error: Failed to allocate\n");
			//exit(0);
		}


		hipMalloc(&d_rc, sar_data_Np * sizeof(float) * sar_data_Nfft*2);
		//printf("point53Cuda Done\n");
		if (hipGetLastError() != hipSuccess) {
			printf("point5233Cuda error: Failed to allocate\n");
			//exit(0);
		}
		/*printf("sardaa_NY111 %d \n", sar_data_Ny);
		dimgridwidth =ceil( sar_data_multiresarray_size/float(sar_data_Ny/RES_SIZEY));
		gridheight = sar_data_Ny /RES_SIZEY;
		
		printf("sardaa_NY1111 %d \n", sar_data_Ny);
		dim3	dimBlock(BLOCK_SIZEX, BLOCK_SIZEY);
		dim3
			dimGrid(dimgridwidth,gridheight );

		printf("sardaa_NY111`11 %d \n", sar_data_Ny);

		//hipFuncSetCacheConfig(reinterpret_cast<const void*>(BPKernel), hipFuncCachePreferL1);
		printf("0 runs %d\n",gpu_id );		
		printf("values %d  %d \n",dimgridwidth, sar_data_Ny );
		printf("sardaa_NY12222 %d \n", sar_data_Ny); */

		hipStream_t stream[totalmpistreamtimes];
		for(int i = 0; i < totalmpistreamtimes; ++i){
		 	hipStreamCreate(&stream[i]);
		}


		

	//}

	//printf("before  alloc 1st point %d   %d \n",gpu_id, mpistreamtimes );		


	

	int dimgridwidth =ceil( sar_data_multiresarray_size/float(sar_data_Ny/RES_SIZEY));
		int gridheight = sar_data_Ny /RES_SIZEY;
		
	
		dim3	dimBlock(BLOCK_SIZEX, BLOCK_SIZEY);
		dim3
			dimGrid(dimgridwidth,gridheight );
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(BPKernel), hipFuncCachePreferL1);

		hipEventRecord(stop8);
		
		hipEventSynchronize(stop8);
		float milliseconds8 = 0;
		hipEventElapsedTime(&milliseconds8, start8, stop8);
		printf("cuda one time  %d gpu  %f  \n",gpu_id, milliseconds8);
	

	
	float totaltimeforcuda=0;
//float totaltimeformpibroadcasting=0;
float totaltimeforpulsearray=0;

	hipEvent_t start4, stop4;
	hipEventCreate(&start4);
	hipEventCreate(&stop4);
	hipEventRecord(start4);

	for(int mpistreamtimes = 0; mpistreamtimes < totalmpistreamtimes; mpistreamtimes++) {

		
/*hipEvent_t start7, stop7;
	hipEventCreate(&start7);
	hipEventCreate(&stop7);
	hipEventRecord(start7); */

	

	/*for ( int i=0; i< sar_data_Np/totalmpistreamtimes; i++){
			

		for ( int j=0; j< sar_data_Nfft; j++){

			
			rc[(sar_data_Np/totalmpistreamtimes) * sar_data_Nfft*mpistreamtimes  +i * sar_data_Nfft + j].x=sar_data_finalIfft_R[(sar_data_Np/totalmpistreamtimes) * sar_data_Nfft*mpistreamtimes  +i * sar_data_Nfft + j];

			
			rc[(sar_data_Np/totalmpistreamtimes) * sar_data_Nfft*mpistreamtimes  +i * sar_data_Nfft + j].y= sar_data_finalIfft_Q[(sar_data_Np/totalmpistreamtimes) * sar_data_Nfft*mpistreamtimes  +i * sar_data_Nfft + j];

		}
	}
     */
				

	/*for (int i = 0; i < sar_data_Np/totalmpistreamtimes; i++) {
		sar_data_Antxyzr[(sar_data_Np/totalmpistreamtimes)*mpistreamtimes +i].x = sar_data_AntX[(sar_data_Np/totalmpistreamtimes)*mpistreamtimes +i];
		sar_data_Antxyzr[(sar_data_Np/totalmpistreamtimes)*mpistreamtimes +i].y = sar_data_AntY[(sar_data_Np/totalmpistreamtimes)*mpistreamtimes +i];
		sar_data_Antxyzr[(sar_data_Np/totalmpistreamtimes)*mpistreamtimes +i].z = sar_data_AntZ[(sar_data_Np/totalmpistreamtimes)*mpistreamtimes +i];
		sar_data_Antxyzr[(sar_data_Np/totalmpistreamtimes)*mpistreamtimes +i].w = sar_data_R0[(sar_data_Np/totalmpistreamtimes)*mpistreamtimes +i];

	} */

	

		

	//	printf("Inside GPu %d Total Number of seconds:  %.10f GPU %d\n",gpu_id, (endset - startset) / (double) CLOCKS_PER_SEC, gpu_id);

		hipMemcpyAsync(&Antxyzr_d[(sar_data_Np/totalmpistreamtimes)*mpistreamtimes], &sar_data_Antxyzr[(sar_data_Np/totalmpistreamtimes)*mpistreamtimes], sizeof(float4) * sar_data_Np/totalmpistreamtimes, hipMemcpyHostToDevice, stream[mpistreamtimes]);

		
	hipMemcpyAsync(&d_rc[(sar_data_Np/totalmpistreamtimes) * sar_data_Nfft*mpistreamtimes], &rc[(sar_data_Np/totalmpistreamtimes) * sar_data_Nfft*mpistreamtimes],
			(sar_data_Np/totalmpistreamtimes) * sizeof(float2) * sar_data_Nfft,
			hipMemcpyHostToDevice,stream[mpistreamtimes]);
	//printf("point54Cuda Done\n");
	if (hipGetLastError() != hipSuccess) {
		printf("point5234Cuda error: Failed to allocate\n");
		//exit(0);
	}
		
		
	
	/*hipEventRecord(stop7);
		
		hipEventSynchronize(stop7);
		float milliseconds7 = 0;
		hipEventElapsedTime(&milliseconds7, start7, stop7);
		totaltimeforpulsearray += milliseconds7; */


		

		//printf(" start working %d \n", gpu_id);

		// kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
		


	BPKernel<<<dimGrid, dimBlock,0,stream[mpistreamtimes]>>>(gpu_id, num_gpus, d_rc, d_im_final_R,d_im_final_Q, sar_data_Nx,sar_data_Ny,sar_data_Nfft,Antxyzr_d,sar_data_minF, sar_data_Np,sar_data_Wx,sar_data_Wy,sar_data_x0,sar_data_y0,prepi,prep2,prep3,sar_data_multiresarray_size,sar_data_multiresarray_offset_size, multiresarray_d,dimgridwidth,totalmpistreamtimes,mpistreamtimes); 

		//hipDeviceSynchronize();

		/*hipEventRecord(stop4);
		hipEventSynchronize(stop4);
		float milliseconds4 = 0;
		hipEventElapsedTime(&milliseconds4, start4, stop4);
		totaltimeforcuda += milliseconds4;*/

		/*hipEvent_t start5, stop5;
		hipEventCreate(&start5);
		hipEventCreate(&stop5);
		hipEventRecord(start5);
*/
		//if(mpistreamtimes<totalmpistreamtimes-1){

		//getbcastarrays(sar_data_xyzr,sar_data_finalIfft_RQ,mpistreamtimes+1,totalmpistreamtimes,gpu_id);

		//}
		

		/*hipEventRecord(stop5);
		hipEventSynchronize(stop5);
		float milliseconds5 = 0;
		hipEventElapsedTime(&milliseconds5, start5, stop5);*/
		//totaltimeformpibroadcasting += milliseconds5;
		
		hipDeviceSynchronize();

		

		//printf(" start working after %d \n", gpu_id);


			//hipDeviceSynchronize();
			//hipStreamSynchronize(stream[mpistreamtimes]);

		//	hipEventRecord(stop1);
		//int  copysize;
		//if(sar_data.Nx >511){
		//	copysize=511;
		//}
		//else{
		//	copysize=sar_data.Nx;
		//}

		
	}
	hipEventRecord(stop4);
		hipEventSynchronize(stop4);
		float milliseconds4 = 0;
		hipEventElapsedTime(&milliseconds4, start4, stop4);
		//totaltimeforcuda += milliseconds4; 
		

	printf("cuda kernel and mpi and memcpy total time: %d gpu  %f  \n",gpu_id, milliseconds4);
	//printf("cuda mpi inside broadcast total time: %d gpu  %f  \n",gpu_id, totaltimeformpibroadcasting);
	//printf("cuda pulse array time %d gpu  %f  \n",gpu_id, totaltimeforpulsearray);

	// measuring time taken




	//hipEvent_t start2, stop2;
	//hipEventCreate(&start2);
	//hipEventCreate(&stop2);
	//hipEventRecord(start2); 






	//int sizeresarray = (sar_data_Nx / RES_SIZE);





	//hipEventRecord(stop2);



	//float min_array_min_value = min_array(sar_data.r_vec, sar_data.Nfft);
	//float max_array_max_value = max_array(sar_data.r_vec, sar_data.Nfft);







	hipEvent_t start1, stop1;
	hipEventCreate(&start1);
	hipEventCreate(&stop1);
	hipEventRecord(start1); 
	//startset2 = clock(); 

	

	

	

	

	//printf(" start working after %d \n", gpu_id);
	//hipDeviceSynchronize();

	

	//if(mpistreamtimes==totalmpistreamtimes-1){


		hipEventRecord(stop1);
		//endset2 = clock();

		//int  copysize;
		//if(sar_data.Nx >511){
		//	copysize=511;
		//}
		//else{
		//	copysize=sar_data.Nx;
		//}

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start);



		hipError_t error;
		error = hipMemcpy(im_final_host_R, d_im_final_R,
				sizeof(float) * (sar_data_multiresarray_size*RES_SIZEX*RES_SIZEY), hipMemcpyDeviceToHost);
		//printf("point55Cuda Done\n");
		if (hipGetLastError() != hipSuccess) {
			printf("CUDA error55: %s in cpu %d %d\n", hipGetErrorString(error),gpu_id, sar_data_multiresarray_size*RES_SIZEX*RES_SIZEY);
			//exit(0);
		}


		error = hipMemcpy(im_final_host_Q, d_im_final_Q,
				sizeof(float) * (sar_data_multiresarray_size*RES_SIZEX*RES_SIZEY), hipMemcpyDeviceToHost);
		//printf("point56Cuda Done\n");
		if (hipGetLastError() != hipSuccess) {
			printf("CUDA error56: %s in cpu %d\n", hipGetErrorString(error),gpu_id);
			//exit(0);
		} 

		



		

		//float milliseconds2 = 0;
		//hipEventElapsedTime(&milliseconds2, start2, stop2);


		//hipEventSynchronize(stop1);
		float milliseconds1 = 0;
		hipEventElapsedTime(&milliseconds1, start1, stop1);


		//printf("Number of total seconds:  %.10f \n", (endset2 - startset2) / (double) CLOCKS_PER_SEC); 





		//printf("CPUID: %d  grid x size %d grid y size %d   \n", gpu_id,dimGrid.x, dimGrid.y  ); 

		//printloop(im_final_host_R,250500,251001);

		/*for (int x = 0; x < 1; x++) {
		  int y;
		  for (y = 0; y < sar_data.Nx; y++) {
		  printf("%lf \n", im_final_host_R[y]);

		  }

		  } */





		hipFree(d_im_final_R);
		hipFree(d_im_final_Q);
		hipFree(Antxyzr_d);
		//hipFree(Anty_d);
		//hipFree(Antz_d);
		//hipFree(r_vec_d);
		//hipFree(R0_d);
		//hipFree(minF_d);
		//hipFree(x_mat_d);
		//hipFree(y_mat_d);
		//hipFree(z_mat_d);
		hipFree(d_rc);

		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);


		hipEventRecord(stop9);
		hipEventSynchronize(stop9);
		float milliseconds9 = 0;
		hipEventElapsedTime(&milliseconds9, start9, stop9);
		//endset = clock();

		//printf("GPU: %d input time: %f \n \n", gpu_id,milliseconds2);

		printf("GPU: %d output image sending time: %f \n ",gpu_id, milliseconds);
		printf("GPU: %d Total  time start to end : %f \n", gpu_id,milliseconds9);

		//printf("Inside GPu %d Total Number of seconds:  %.10f GPU %d\n",gpu_id, (endset - startset) / (double) CLOCKS_PER_SEC, gpu_id);

	//}

	//free(final_im_final);

}


__device__ hipfftComplex complexexp(hipfftComplex z)

{

	float factor = exp(z.x);
	return make_hipFloatComplex(factor * cosf(z.y), factor * sinf(z.y));

}

void signalIFFT(hipfftComplex *d_signal, int signal_size) {

	hipfftHandle plan;
	if (hipfftPlan1d(&plan, signal_size, HIPFFT_C2C, 1) != HIPFFT_SUCCESS) {
		printf("Failed to plan IFFT\n");
		exit(0);
	}

	// Execute the plan.
	if (hipfftExecC2C(plan, (hipfftComplex *) d_signal, (hipfftComplex *) d_signal,
				HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
		printf("Failed Executing IFFT\n");
		exit(0);
	}

	hipfftDestroy(plan);	

}

void getIfftresults(int sar_data_Np, int sar_data_Nfft, int sar_data_K, float** sar_data_pdcph_I, float** sar_data_pdcph_Q, float* sar_data_finalIfft_RQ ){
	int i;






	for (int li = 0; li < sar_data_Np; li++) {
		hipfftComplex *d_signal1;
		hipMalloc(&d_signal1, sizeof(hipfftComplex) * sar_data_Nfft);

		if (hipGetLastError() != hipSuccess) {
			printf("pointnewnew1Cuda error: Failed to allocate\n");
			exit(0);
		}
		hipfftComplex rc[4096];

		for (i = 0; i < sar_data_Nfft; i++) {

			if (i < sar_data_K) {
				rc[i].x = sar_data_pdcph_I[li][i];

				rc[i].y = sar_data_pdcph_Q[li][i];
			} else {
				rc[i].x = 0;

				rc[i].y = 0;

			}

		}

		hipMemcpy(d_signal1, rc, sizeof(hipfftComplex) * sar_data_Nfft,
				hipMemcpyHostToDevice);

		if (hipGetLastError() != hipSuccess) {
			printf("point51Cuda error: Failed to allocate\n");
			exit(0);
		}

		signalIFFT(d_signal1, sar_data_Nfft);
		hipDeviceSynchronize();
		hipMemcpy(rc, d_signal1, sizeof(hipfftComplex) * sar_data_Nfft,
				hipMemcpyDeviceToHost);

		if (hipGetLastError() != hipSuccess) {
			printf("point52Cuda error: Failed to allocate\n");
			exit(0);
		}
		for (i = 0; i < sar_data_Nfft; i++) {

			rc[i].x = (rc[i].x) / sar_data_Nfft;
			rc[i].y = (rc[i].y) / sar_data_Nfft;

		}

		int cswap = (int) sar_data_Nfft / 2;

		hipfftComplex tmp;

		for (int k = 0; k < cswap; k++) {
			tmp = rc[k];
			rc[k] = rc[k + cswap];
			rc[k + cswap] = tmp;

		}

		for (i = 0; i < sar_data_Nfft; i++) {

			sar_data_finalIfft_RQ[(li * sar_data_Nfft + i)*2+0] = rc[i].x;
			sar_data_finalIfft_RQ[(li * sar_data_Nfft + i)*2+1] = rc[i].y;

		}

		hipFree(d_signal1);
		//printf("working here %d \n",li);
		//li++;

	}  


}

int getDeviceCount(){
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	return deviceCount;

}


